/* 
Problem 4A): CUDA program to initialize an array array[] of size 8000 to all 2s in parallel.
    # Akanksh Jagadish 885199208
    # Jorge Mejia 888533239,
    # Samee Rauf 888657053
    # Hamid Suha 886960848
// compile with: // nvcc -o problem4A problem4A.cu
*/

#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 8000

__global__ void add(int *a, int *b) {
    int i = blockIdx.x;
    if (i < N) {
        b[i] = 2;
}
}
int main() {
    // cpu array        
    int ha[N], hb[N];

    //device array
    int *da, *db;
    hipMalloc((void **)&da, N*sizeof(int));
    hipMalloc((void **)&db, N*sizeof(int));

    //initialize cpu array
    for (int i = 0; i < N; ++i){
            ha[i] = i;
        }
    // transfer input from cpu to device
    hipMemcpy(da, ha, N*sizeof(int), hipMemcpyHostToDevice);

    // launch GPU code

    add <<<N, 1 >>>(da, db);

    // copy output array from GPU back to CPU

   hipMemcpy(hb, db, N*sizeof(int), hipMemcpyDeviceToHost);

   for(int i = 0; i < N; ++i) {
        printf("element: %d : %d\n",i, hb[i]);
   }
   hipFree(da);
   hipFree(db);
    return 0;
}
