/* 
Problem 4B): Create another kernel that adds i to array[i] with a size of 8000
    # Akanksh Jagadish 885199208
    # Jorge Mejia 888533239,
    # Samee Rauf 888657053
    # Hamid Suha 886960848
// compile with: // nvcc -o problem4B problem4B.cu
*/

#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 8000

__global__ void initialize(int *a, int *b) {
    int i = blockIdx.x;
    if (i < N) {
        b[i] = 2;
        }
}
__global__ void add(int *a, int *b) {
    int i = blockIdx.x;
    if (i < N) {
        b[i] = b[i] + i;
        }
}
int main() {
    // cpu array        
    int ha[N], hb[N];

    //device array
    int *da, *db;
    hipMalloc((void **)&da, N*sizeof(int));
    hipMalloc((void **)&db, N*sizeof(int));

    //initialize cpu array
    for (int i = 0; i < N; ++i){
            ha[i] = i;
        }
    // transfer input from cpu to device
    hipMemcpy(da, ha, N*sizeof(int), hipMemcpyHostToDevice);

    // launch GPU code

    initialize <<<N, 1 >>>(da, db);

    // copy output array from GPU back to CPU

   hipMemcpy(hb, db, N*sizeof(int), hipMemcpyDeviceToHost);

   for(int i = 0; i < N; ++i) {
        printf("element: %d : %d\n",i, hb[i]);
   }

   add <<< N,1 >>>(da, db);
   hipMemcpy(hb, db, N*sizeof(int), hipMemcpyDeviceToHost);
   printf("\nvaleues of array after adding i : \n");
   for(int i = 0; i < N; ++i) {
        printf("element: %d : %d\n",i, hb[i]);
   }
   hipFree(da);
   hipFree(db);
    return 0;
}
