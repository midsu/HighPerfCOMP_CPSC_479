                                                                                                                                                                   #include <iostream>
#include <hip/hip_runtime.h>

__device__ void heapify(int *arr, int n, int i) {
    int largest = i;
        int left = 2 * i + 1;
            int right = 2 * i + 2;

    if (left < n && arr[left] > arr[largest]) {
            largest = left;
                }

    if (right < n && arr[right] > arr[largest]) {
            largest = right;
                }

    if (largest != i) {
            // Swap arr[i] and arr[largest]
                    int temp = arr[i];
                            arr[i] = arr[largest];
                                    arr[largest] = temp;

        // Recursively heapify the affected sub-tree
                heapify(arr, n, largest);
                    }
                    }

__global__ void heapSort(int *arr, int n) {
    // Build max heap
        for (int i = n / 2 - 1; i >= 0; i--) {
                heapify(arr, n, i);
                    }

    // Extract elements one by one from the heap
        for (int i = n - 1; i >= 0; i--) {
                // Move current root to the end
                        int temp = arr[0];
                                arr[0] = arr[i];
                                        arr[i] = temp;

        // Call max heapify on the reduced heap
                heapify(arr, i, 0);
                    }
                    }
int main() {
    const int arraySize = 8;
        int h_array[arraySize] = {25, 33, 70, 102, 56, 9, 11, 32};
            int *d_array;
        std::cout << "Unsorted array: ";
                  for (int j = 0; j < arraySize; j++){
                  std::cout << h_array[j] << " ";
                  }
        std::cout << "\n";
    // Allocate device memory
        hipMalloc((void**)&d_array, arraySize * sizeof(int));

    // Copy data from host to device
        hipMemcpy(d_array, h_array, arraySize * sizeof(int), hipMemcpyHostToDevice);
// Example: Launch with 256 threads per block and a grid size based on array size
        int blockSize = 256;
            int gridSize = (arraySize + blockSize - 1) / blockSize;
                heapSort<<<gridSize, blockSize>>>(d_array, arraySize);

    // Copy the result back to the host
        hipMemcpy(h_array, d_array, arraySize * sizeof(int), hipMemcpyDeviceToHost);

    // Free device memory
        hipFree(d_array);

    // Print sorted array
        std::cout << "Sorted Array: ";
            for (int i = 0; i < arraySize; i++) {
                    std::cout << h_array[i] << " ";
                        }
                            std::cout << std::endl;

    return 0;
    }
