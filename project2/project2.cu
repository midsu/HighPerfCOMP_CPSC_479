
/*************************************************
 *  
 *  CPSC 479 Project 2: Introduction to HPC - Data Science project
 * 
 *  Student: Hamid Suha
 *  Instructor: Prof. Doina Bein
 * 
 * 
 *  Parallel sorting - Heap Sort
 * 
 *  TO COMPILE AND RUN
 *      nvcc proj2.cu
 *      ./a.out 
 * 
 * 
**************************************************/

#include <iostream>
#include <hip/hip_runtime.h>

__device__ void heapify(int *arr, int n, int i) {
    int largest = i;
    int left = 2 * i + 1;
    int right = 2 * i + 2;

    if (left < n && arr[left] > arr[largest]) {
        largest = left;
    }

    if (right < n && arr[right] > arr[largest]) {
        largest = right;
    }

    if (largest != i) {
        // Swap arr[i] and arr[largest]
        int temp = arr[i];
        arr[i] = arr[largest];
        arr[largest] = temp;

        // Recursively heapify the affected sub-tree
        heapify(arr, n, largest);
    }
}

__global__ void heapSort(int *arr, int n) {
    // Build max heap
    for (int i = n / 2 - 1; i >= 0; i--) {
        heapify(arr, n, i);
    }

    // Extract elements one by one from the heap
    for (int i = n - 1; i >= 0; i--) {
        // Move current root to the end
        int temp = arr[0];
        arr[0] = arr[i];
        arr[i] = temp;

        // Call max heapify on the reduced heap
        heapify(arr, i, 0);
    }
}

int main() {
    const int arraySize = 10;
    int h_array[arraySize] = {12, 11, 13, 5, 6, 7, 2, 9, 8, 10};
    int *d_array;

    // Allocate device memory
    hipMalloc((void**)&d_array, arraySize * sizeof(int));

    // Copy data from host to device
    hipMemcpy(d_array, h_array, arraySize * sizeof(int), hipMemcpyHostToDevice);

    // Example: Launch with 256 threads per block and a grid size based on array size
    int blockSize = 256;
    int gridSize = (arraySize + blockSize - 1) / blockSize;
    heapSort<<<gridSize, blockSize>>>(d_array, arraySize);

    // Copy the result back to the host
    hipMemcpy(h_array, d_array, arraySize * sizeof(int), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_array);

    // Print sorted array
    std::cout << "Sorted Array: ";
    for (int i = 0; i < arraySize; i++) {
        std::cout << h_array[i] << " ";
    }
    std::cout << std::endl;

    return 0;
}

/*************************************************
 *  
 *  Sample Output
 *  
 *  test case 1:
 *  array: {12, 11, 13, 5, 6, 7, 2, 9, 8, 10}
 *  sorted array: {2, 5, 6, 7, 8, 9, 10, 11, 12}
 * 
 *  test case 2:
 *  array: {40, 1, 12, 11, 22, 4, 13, 5, 6, 7, 2, 9, 8, 10}
 *  sorted array: 0 1 2 4 5 6 7 8 9 10 11 12 13 22 40 
 * 
 * 
**************************************************/

