/* 
Excercise 2: Compute matrix square
    # Akanksh Jagadish 885199208
    # Jorge Mejia 888533239,
    # Samee Rauf 888657053
    # Hamid Suha 886960848
// compile with: // nvcc -o ex2 excercise2.cu
*/
#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 16

__global__ void square(unsigned *result, unsigned *matrix, unsigned matrixsize) {
    unsigned id = blockIdx.x * blockDim.x + threadIdx.x;

    unsigned ii = id / matrixsize;
    unsigned jj = id % matrixsize;

    for (unsigned kk = 0; kk < matrixsize; ++kk) {
        result[ii * matrixsize + jj] += matrix[ii * matrixsize + kk] * matrix[kk * matrixsize + jj];
    }
}

int main() {
    dim3 block(N * N, 1, 1);

    unsigned *matrix, *hmatrix, *result;

    hipMalloc(&matrix, N * N * sizeof(unsigned));
    hipMalloc(&result, N * N * sizeof(unsigned));

    hmatrix = (unsigned *)malloc(N * N * sizeof(unsigned));
    printf("Matrix elements:\n");
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            hmatrix[i * N + j] = 2;
            printf("%u ", hmatrix[i * N + j]);
        }
        printf("\n");
    }

    hipMemcpy(matrix, hmatrix, N * N * sizeof(unsigned), hipMemcpyHostToDevice);

    square<<<1, block>>>(result, matrix, N);

    hipMemcpy(hmatrix, result, N * N * sizeof(unsigned), hipMemcpyDeviceToHost);

    printf("Result matrix after squaring:\n");
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            printf("%u ", hmatrix[i * N + j]);
        }
        printf("\n");
    }

    // Free allocated memory on the device
    hipFree(matrix);
    hipFree(result);

    // Free allocated memory on the host
    free(hmatrix);

    hipDeviceSynchronize();

    return 0;
}
