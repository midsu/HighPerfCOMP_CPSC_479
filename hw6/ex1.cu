/* 
Excercise 1 : compute even values from 2d matrix
    # Akanksh Jagadish 885199208
    # Jorge Mejia 888533239,
    # Samee Rauf 888657053
    # Hamid Suha 886960848
// compile with: // nvcc -o ex1 ex1.cu
*/
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void dkernel(int *solution, unsigned *matrix) {
    unsigned id = threadIdx.x * blockDim.y + threadIdx.y;
    if (matrix[id] % 2 == 0) {
        atomicAdd(solution, 1);
    }
}

#define N 16

int main() {
    dim3 block(N, N, 1);

    unsigned *matrix, *hmatrix;
    int *solution;
    int *hsolution = (int *)malloc(sizeof(int));
    *hsolution = 0;

    hipMalloc(&matrix, N * N * sizeof(unsigned));
    hipMalloc(&solution, sizeof(int));

    hmatrix = (unsigned *)malloc(N * N * sizeof(unsigned));
    printf("Matrix elements:\n");
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            hmatrix[i * N + j] = 2;
            printf("%u ", hmatrix[i * N + j]);
        }
        printf("\n");
    }

    hipMemcpy(matrix, hmatrix, N * N * sizeof(unsigned), hipMemcpyHostToDevice);
    hipMemcpy(solution, hsolution, sizeof(int), hipMemcpyHostToDevice);

    dkernel<<<1, block>>>(solution, matrix);

    hipMemcpy(hsolution, solution, sizeof(int), hipMemcpyDeviceToHost);

    printf("Number of even elements: %d\n", *hsolution);

    hipDeviceSynchronize();
    return 0;
}
